
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define SPHERES 20
#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

struct Sphere
{
    float r, b, g;
    float radius;
    float x, y, z;
};

__device__ float hit(float ox, float oy, const Sphere *s, float *n)
{
    float dx = ox - s->x;
    float dy = oy - s->y;
    if (dx * dx + dy * dy < s->radius * s->radius)
    {
        float dz = sqrtf(s->radius * s->radius - dx * dx - dy * dy);
        *n = dz / sqrtf(s->radius * s->radius);
        return dz + s->z;
    }
    return -INF;
}

__global__ void rayTracingKernel(Sphere *s_dev, unsigned char *bitmap_dev)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < DIM && y < DIM)
    {
        int offset = x + y * DIM;
        float ox = (x - DIM / 2.0f);
        float oy = (y - DIM / 2.0f);

        float r = 0, g = 0, b = 0;
        float maxz = -INF;
        for (int i = 0; i < SPHERES; i++) {
            float n;
            float t = hit(ox, oy, &s_dev[i], &n);
            if (t > maxz) {
                float fscale = n;
                r = s_dev[i].r * fscale;
                g = s_dev[i].g * fscale;
                b = s_dev[i].b * fscale;
                maxz = t;
            }
        }

        bitmap_dev[offset * 4 + 0] = (unsigned char)(r * 255);
        bitmap_dev[offset * 4 + 1] = (unsigned char)(g * 255);
        bitmap_dev[offset * 4 + 2] = (unsigned char)(b * 255);
        bitmap_dev[offset * 4 + 3] = 255;
    }
}

void ppm_write(unsigned char *bitmap, int xdim, int ydim, FILE *fp)
{
    int i, x, y;
    fprintf(fp, "P3\n");
    fprintf(fp, "%d %d\n", xdim, ydim);
    fprintf(fp, "255\n");
    for (y = 0; y < ydim; y++) {
        for (x = 0; x < xdim; x++) {
            i = x + y * xdim;
            fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
        }
        fprintf(fp, "\n");
    }
}

int main(int argc, char *argv[])
{
    Sphere *s_host;
    Sphere *s_dev;
    unsigned char *bitmap_host;
    unsigned char *bitmap_dev;

    hipEvent_t start, stop;
    float milliseconds = 0;

    srand(time(NULL));

    s_host = (Sphere *)malloc(sizeof(Sphere) * SPHERES);
    if (!s_host) {
        perror("Failed to allocate host memory for spheres");
        return 1;
    }

    for (int i = 0; i < SPHERES; i++) {
        s_host[i].r = rnd(1.0f);
        s_host[i].g = rnd(1.0f);
        s_host[i].b = rnd(1.0f);
        s_host[i].x = rnd(2000.0f) - 1000;
        s_host[i].y = rnd(2000.0f) - 1000;
        s_host[i].z = rnd(2000.0f) - 1000;
        s_host[i].radius = rnd(200.0f) + 40;
    }

    // Allocate device memory
    hipMalloc((void**)&s_dev, sizeof(Sphere) * SPHERES);
    hipMalloc((void**)&bitmap_dev, sizeof(unsigned char) * DIM * DIM * 4);

    // Copy sphere data from host to device
    hipMemcpy(s_dev, s_host, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks( (DIM + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (DIM + threadsPerBlock.y - 1) / threadsPerBlock.y );

    hipEventRecord(start); // Start timing

    // Launch CUDA kernel
    rayTracingKernel<<<numBlocks, threadsPerBlock>>>(s_dev, bitmap_dev);

    hipEventRecord(stop);  // End timing
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Allocate host memory for the result bitmap
    bitmap_host = (unsigned char *)malloc(sizeof(unsigned char) * DIM * DIM * 4);
    if (!bitmap_host) {
        perror("Failed to allocate host memory for bitmap");
        hipFree(s_dev);
        hipFree(bitmap_dev);
        free(s_host);
        return 1;
    }

    // Copy result bitmap from device to host
    hipMemcpy(bitmap_host, bitmap_dev, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost);

    FILE *fp = fopen("result.ppm", "w");
    if (!fp) {
        perror("Failed to open result.ppm for writing");
        free(bitmap_host);
        free(s_host);
        hipFree(s_dev);
        hipFree(bitmap_dev);
        return 1;
    }

    ppm_write(bitmap_host, DIM, DIM, fp);
    fclose(fp);

    printf("CUDA ray tracing: %.3f sec\n", milliseconds / 1000.0f); // Convert ms to sec
    printf("[result.ppm] was generated.\n");

    free(s_host);
    free(bitmap_host);
    hipFree(s_dev);
    hipFree(bitmap_dev);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
